
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * A simple introduction to programming in CUDA. 
 * This program prints "Hello World from GPU! from 10 CUDA threads running on the GPU."
 * (see - https://github.com/deeperlearning/professional-cuda-c-programming )
 */

__global__ void helloFromGPU()
{
    printf("Hello World from GPU!\n");
}

int main(int argc, char **argv)
{
    printf("Hello World from CPU!\n");

    helloFromGPU<<<1, 10>>>();
    hipDeviceReset();
    return 0;
}
